
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<cmath>
using namespace std;

#define Lx 16
#define Nx 8
const int Mx = (Lx+Nx-1)/Nx;

//-------- Device's Program-----
//-------- Kernels--------------

__global__ void AddTwoVectors(float* d_a, float* d_b, float* d_c){
	//Who am I?
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	d_c[ix] = d_a[ix] + d_b[ix];
}
//-------- Host's Program-----
int main(void){
	int ix;
	//Declara variables por duplicado
	//---DEVICE---
	float *d_a; hipMalloc((void**) &d_a, Lx*sizeof(float));
	float *d_b; hipMalloc((void**) &d_b, Lx*sizeof(float));
	float *d_c; hipMalloc((void**) &d_c, Lx*sizeof(float));
	//---HOST-----
	float h_a[Lx], h_b[Lx], h_c[Lx];

	//Fill data for proccesing

	for (ix =0; ix < Lx; ix++){
		h_a[ix]=ix; 
		h_b[ix]=2*ix; 
	}

	//Send to Device
	hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(d_b,h_b,Lx*sizeof(float),hipMemcpyHostToDevice);

	//Run in Device
	dim3 BlocksPerGrid(Mx,1,1);
	dim3 ThreadsPerBlock(Nx,1,1);
	AddTwoVectors<<<BlocksPerGrid, ThreadsPerBlock >>>(d_a,d_b,d_c);
	//Return result to Host
	hipMemcpy(h_c, d_c,Lx*sizeof(float), hipMemcpyDeviceToHost);
	//Print
		for(ix=0; ix<Lx; ix++){
		cout << h_a[ix] << "\t" << h_b[ix] << "\t" << h_c[ix] << "\n";
		}
	//Free Dynamic memory
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}
