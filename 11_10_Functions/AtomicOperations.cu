
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
using namespace std;

#define Lx 16
#define Nx 8
const int Mx=(Lx+Nx-1)/Nx;

//--------------- KERNELS ----------------
__global__ void SumarTodos(float *d_a,float *d_Resultado){

  //cargar los datos a la memoria compartida

  int ix; ix=blockIdx.x*blockDim.x+threadIdx.x;

  //d_Resultado[0]+=d_a[ix];
  atomicAdd(&d_Resultado[0],d_a[ix]);

}

int main(void){

  int ix;
  //DECLARAR LAS MATRICES
  float h_a[Lx];         float *d_a;  hipMalloc((void**) &d_a,Lx*sizeof(float));
  float h_Resultado[1];  float *d_Resultado;  hipMalloc((void**) &d_Resultado,sizeof(float));

  //INICIALIZAR LOS DATOS

  //Cargarlos en el Host

  for(ix=0;ix<Lx;ix++){
    h_a[ix]=ix;
  }

  h_Resultado[0]=0;

  //IMPRIMIRLOS

  for(ix=0;ix<Lx;ix++)
    cout<<h_a[ix]<<" ";
  cout<<endl;


  //Enviarlos al Device
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Resultado,h_Resultado,sizeof(float),hipMemcpyHostToDevice);

  //PROCESAR EN LA TARJETA GRAFICA

  dim3 ThreadsPerBlock(Nx,1,1);
  dim3 BlocksPerGrid(Mx,1,1);

  SumarTodos<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a,d_Resultado);

  //DEVOLVERLOS AL HOST
  hipMemcpy(h_Resultado,d_Resultado,sizeof(float),hipMemcpyDeviceToHost);

  //IMPRIMIRLOS
  cout<<"EL resultado es ="<<h_Resultado[0]<<endl;

  //LIBERAR MEMORIA
  hipFree(d_a);  hipFree(d_Resultado);

  return 0;
}
