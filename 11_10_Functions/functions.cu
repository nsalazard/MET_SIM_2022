
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
using namespace std;

#define Lx 16
#define Nx 8
const int Mx=(Lx+Nx-1)/Nx;

//--------------- KERNELS ----------------
__global__ void SumeDeATres(float *d_a){
  extern __shared__ float Shared[];
  //cargar los datos a la memoria compartida
  int ix; ix=blockIdx.x*blockDim.x+threadIdx.x;
  int is; is=threadIdx.x;
  //Cargar la memoria compartida
  switch(is){
  case 0           : Shared[is+1]=d_a[ix]; Shared[0]=d_a[(ix-1+Lx)%Lx]; break;
  case Nx-1        : Shared[is+1]=d_a[ix]; Shared[is+2]=d_a[(ix+1)%Lx]; break;
  default          : Shared[is+1]=d_a[ix];
  }
  //Sincronizar
  __syncthreads();
  //Hacer la suma y escribir
  int Left=is,   Center=is+1,  Right=is+2;
  d_a[ix]=Shared[Left]+Shared[Center]+Shared[Right];
}

int main(void){
  int ix;
  //DECLARAR LAS MATRICES
  float h_a[Lx];
  float *d_a;  hipMalloc((void**) &d_a,Lx*sizeof(float));

 //INICIALIZAR LOS DATOS
  //Cargarlos en el Host
  for(ix=0;ix<Lx;ix++){
    h_a[ix]=ix;
  }

 //IMPRIMIRLOS
  for(ix=0;ix<Lx;ix++)
    cout<<h_a[ix]<<" ";
  cout<<endl;

 //Enviarlos al Device
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);

  //PROCESAR EN LA TARJETA GRAFICA
  dim3 ThreadsPerBlock(Nx,1,1);
  dim3 BlocksPerGrid(Mx,1,1);
  SumeDeATres<<<BlocksPerGrid,ThreadsPerBlock, (Nx+2)*sizeof(float)>>>(d_a);

 //DEVOLVERLOS AL HOST
  hipMemcpy(h_a,d_a,Lx*sizeof(float),hipMemcpyDeviceToHost);

 //IMPRIMIRLOS
  for(ix=0;ix<Lx;ix++)
    cout<<h_a[ix]<<" ";
  cout<<endl;

 //LIBERAR MEMORIA
  hipFree(d_a);
	
 return 0;
}
