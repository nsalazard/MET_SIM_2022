
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
using namespace std;



#define Lx 16
#define Nx 8
const int Mx=(Lx+Nx-1)/Nx;



//--------------- KERNELS ----------------
__global__ void ReduceThreads(float * d_a,float * d_ReduceResult){
  //Set global index
  int ix=threadIdx.x+blockIdx.x*blockDim.x;
  //Define and load shared memory
  extern __shared__ unsigned int temp[];
  temp[threadIdx.x]=d_a[ix];
  __syncthreads();
  //Reduce loop
  for(unsigned int s=blockDim.x/2;s>0;s>>=1){
    if(threadIdx.x<s)
      temp[threadIdx.x]+=temp[threadIdx.x+s];
    __syncthreads();
  }
  //Write to d_ReduceResult
  switch(threadIdx.x){
  case 0: d_ReduceResult[blockIdx.x]=temp[0]; break;
  default: ;
  }
}



int main(){
  //DECLARAR LAS VARIABLES
  int ix,bx;
  //DECLARAR LAS MATRICES
  //En el Host
  float h_a[Lx],h_ReduceResult[Mx],h_TotalResult[1];
  //En el Device
  float *d_a;             hipMalloc((void**) &d_a,Lx*sizeof(float));
  float *d_ReduceResult;  hipMalloc((void**) &d_ReduceResult,Mx*sizeof(float));
  float *d_TotalResult;   hipMalloc((void**) &d_TotalResult,sizeof(float));



 //INICIALIZAR LOS DATOS
  //Cargarlos en el Host
  for(ix=0;ix<Lx;ix++) h_a[ix]=ix;
  for(bx=0;bx<Mx;bx++) h_ReduceResult[bx]=0;
  h_TotalResult[0]=0;
  //Enviarlos al Device
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ReduceResult,h_ReduceResult,Mx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_TotalResult,h_TotalResult,Mx*sizeof(float),hipMemcpyHostToDevice);
  //Imprimirlos
  for(ix=0;ix<Lx;ix++) cout<<h_a[ix]<<" "; cout<<endl;
  
  //SUMAR DENTRO DE CADA BLOQUE
  //Procesar en la tarjeta gráfica
  dim3 ThreadsPerBlock1(Nx,1,1);
  dim3 BlocksPerGrid1(Mx,1,1);
  ReduceThreads<<<BlocksPerGrid1,ThreadsPerBlock1,Nx*sizeof(float)>>>(d_a,d_ReduceResult);
  //Imprimir los resultados parciales
  hipMemcpy(h_ReduceResult,d_ReduceResult,Mx*sizeof(float),hipMemcpyDeviceToHost);
  for(bx=0;bx<Mx;bx++) cout<<h_ReduceResult[bx]<<" "; cout<<endl;
  
  //SUMAR LAS SUMAS PARCIALES DE LOS BLOQUES EN UN SOLO DATO
  //Procesar en la tarjeta gráfica
  dim3 ThreadsPerBlock2(Mx,1,1);
  dim3 BlocksPerGrid2(1,1,1);
  ReduceThreads<<<BlocksPerGrid2,ThreadsPerBlock2,Mx*sizeof(float)>>>(d_ReduceResult,d_TotalResult);
  //Imprimir los resultados parciales
  hipMemcpy(h_TotalResult,d_TotalResult,sizeof(float),hipMemcpyDeviceToHost);
  cout<<h_TotalResult[0]<<endl;



 //LIBERAR MEMORIA
  hipFree(d_a);  hipFree(d_ReduceResult);  hipFree(d_TotalResult);



 return 0;
}
